#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// A basic kernel
__global__ void kernel_test(float* A)
{
    A[threadIdx.x] = (float)(threadIdx.x);
    
    // Add a voluntary error to test colornvcc
    A[0.1] = 42;
}

int main()
{
    const unsigned int NTHREADS = 32;
    const unsigned int NBLOCKS = 64;
    
    printf("Hello World!\n");

    float h_A[NTHREADS];
    int memsize = NTHREADS * sizeof(float);
    memset(&h_A, 0, memsize);

    float* d_A;
    
    // Allocate and copy data to the device
    hipMalloc((void**)&d_A, memsize);
    hipMemcpy(d_A, h_A, memsize, hipMemcpyHostToDevice);

    // Run the kernel
    kernel_test<<<NBLOCKS, NTHREADS>>>(d_A);

    // Copy data back to the host
    hipMemcpy(h_A, d_A, memsize, hipMemcpyDeviceToHost);
    hipFree(d_A);
}
